#include "hip/hip_runtime.h"
#include "yolo.h"
#include "process.cuh"

#include <onnx-tensorrt/NvOnnxParser.h>
#include <hip/hip_runtime.h>

// ref: https://cocodataset.org/#home
const char* cocolabels[] = {
    "person", "bicycle", "car", "motorcycle", "airplane",
    "bus", "train", "truck", "boat", "traffic light", "fire hydrant",
    "stop sign", "parking meter", "bench", "bird", "cat", "dog", "horse",
    "sheep", "cow", "elephant", "bear", "zebra", "giraffe", "backpack",
    "umbrella", "handbag", "tie", "suitcase", "frisbee", "skis",
    "snowboard", "sports ball", "kite", "baseball bat", "baseball glove",
    "skateboard", "surfboard", "tennis racket", "bottle", "wine glass",
    "cup", "fork", "knife", "spoon", "bowl", "banana", "apple", "sandwich",
    "orange", "broccoli", "carrot", "hot dog", "pizza", "donut", "cake",
    "chair", "couch", "potted plant", "bed", "dining table", "toilet", "tv",
    "laptop", "mouse", "remote", "keyboard", "cell phone", "microwave",
    "oven", "toaster", "sink", "refrigerator", "book", "clock", "vase",
    "scissors", "teddy bear", "hair drier", "toothbrush"
};


class YoloImpl : public Yolo {
public:    
    YoloImpl(char* onnxFile,
             char* engineFile,
             bool useInt8,
             int input_batch,
             int input_channel,
             int input_height,
             int input_width,
             TRTLogger* logger);
    ~YoloImpl();

public:
    void detect(cv::Mat& image, std::vector<DetectBox>& allDetections) override;
    bool build_model();
    bool load_model();

private:
    void preprocess(const std::string& device, float* d2i, cv::Mat& image);
    void postprocess(const std::string& device,
                     const int& output_numel,
                     const int& output_numbox,
                     float* output_data_device,
                     const int& num_classes,
                     float* d2i,
                     std::vector<DetectBox>& allDetections);
    void postprocess_cpu(const int& output_numel,
                         const int& output_numbox,
                         float* output_data_device,
                         const int& num_classes,
                         float* d2i,
                         std::vector<DetectBox>& allDetections);
    void postprocess_gpu(const int& output_numbox,
                         float* output_data_device,
                         const int& num_classes,
                         float* d2i,
                         std::vector<DetectBox>& allDetections);

private:
    const char* onnxFile;
    const char* engineFile;
    bool useInt8;

    int input_batch;
    int input_channel;
    int input_height;
    int input_width;
    int input_numel;
    float* input_data_host;
    float* input_data_device;

    int output_numbox;
    int output_numprob;
    int num_classes;
    int output_numel;
    float* output_data_device;

    float* bindings[2];

private:
    std::shared_ptr<nvinfer1::ICudaEngine> engine;
    std::shared_ptr<nvinfer1::IExecutionContext> execution_context;
    hipStream_t stream;
    TRTLogger* logger;
};


YoloImpl::YoloImpl(char* onnxFile, char* engineFile, bool useInt8, int input_batch, int input_channel, int input_height, int input_width, TRTLogger* logger) {
    this->onnxFile = onnxFile;
    this->engineFile = engineFile;
    this->useInt8 = useInt8;
    this->input_batch = input_batch;
    this->input_channel = input_channel;
    this->input_height = input_height;
    this->input_width = input_width;
    this->input_numel = input_batch * input_channel * input_height * input_width;
    this->logger = logger;
}


YoloImpl::~YoloImpl() {
    if (stream != nullptr) checkRuntime(hipStreamDestroy(stream));
    if (input_data_host != nullptr) checkRuntime(hipHostFree(input_data_host));
    if (input_data_device != nullptr) checkRuntime(hipFree(input_data_device));
    if (output_data_device != nullptr) checkRuntime(hipFree(output_data_device));
}


// build yolov5s engine, save to workspace/engineFile
bool YoloImpl::build_model() {

    if(exists(this->engineFile)){
        printf("%s has exists.\n", this->engineFile);
        return true;
    } else {
        printf("Building %s ...\n", this->engineFile);
    }

    // make_nvshared, destroy automatically
    auto builder = make_nvshared(nvinfer1::createInferBuilder(*logger));
    auto config = make_nvshared(builder->createBuilderConfig());
    auto network = make_nvshared(builder->createNetworkV2(1));

    // parse network data from onnx file to `network`
    auto parser = make_nvshared(nvonnxparser::createParser(*network, *logger));
    if(!parser->parseFromFile(this->onnxFile, 1)){
        printf("Failed to parse %s\n", this->onnxFile);
        return false;
    }
    
    int maxBatchSize = 10;
    printf("Workspace Size = %.2f MB\n", (1 << 28) / 1024.0f / 1024.0f);
    config->setMaxWorkspaceSize(1 << 28);

    auto profile = builder->createOptimizationProfile();
    auto input_tensor = network->getInput(0);
    auto input_dims = input_tensor->getDimensions();
    
    // configure minimum, optimal, and maximum ranges
    input_dims.d[0] = 1;

    if (this->useInt8) {
        config->setFlag(nvinfer1::BuilderFlag::kINT8);
    }

    profile->setDimensions(input_tensor->getName(), nvinfer1::OptProfileSelector::kMIN, input_dims);
    profile->setDimensions(input_tensor->getName(), nvinfer1::OptProfileSelector::kOPT, input_dims);
    input_dims.d[0] = maxBatchSize;
    profile->setDimensions(input_tensor->getName(), nvinfer1::OptProfileSelector::kMAX, input_dims);
    config->addOptimizationProfile(profile);

    auto engine = make_nvshared(builder->buildEngineWithConfig(*network, *config));
    if(engine == nullptr){
        printf("Build %s failed.\n", this->engineFile);
        return false;
    }

    // serialize model to engine file
    auto model_data = make_nvshared(engine->serialize());
    FILE* f = fopen(this->engineFile, "wb");
    fwrite(model_data->data(), 1, model_data->size(), f);
    fclose(f);

    printf("Build done.\n");
    return true;
}


bool YoloImpl::load_model() {

    auto yolov5_engine_data = load_file(this->engineFile);
    auto runtime   = make_nvshared(nvinfer1::createInferRuntime(*(this->logger)));
    this->engine = make_nvshared(runtime->deserializeCudaEngine(yolov5_engine_data.data(), yolov5_engine_data.size()));
    if(engine == nullptr){
        printf("Deserialize cuda engine failed.\n");
        runtime->destroy();
        return false;
    }
    if (engine->getNbBindings() != 2) {
        printf("ONNX export error: Must have exactly 1 input and "
            "1 output, but you have: %d outputs.\n", 
            engine->getNbBindings() - 1);
        return false;
    }
    this->stream = nullptr;
    checkRuntime(hipStreamCreate(&this->stream));
    this->execution_context = make_nvshared(engine->createExecutionContext());
    checkRuntime(hipHostMalloc(&this->input_data_host, this->input_numel * sizeof(float)));
    checkRuntime(hipMalloc(&this->input_data_device, this->input_numel * sizeof(float)));

    // 3x3 input -> 3x3 output
    auto output_dims = engine->getBindingDimensions(1);
    this->output_numbox = output_dims.d[1];
    this->output_numprob = output_dims.d[2];
    this->num_classes = output_numprob - 5;
    this->output_numel = input_batch * output_numbox * output_numprob;

    this->output_data_device = nullptr;
    checkRuntime(hipMalloc(&this->output_data_device, sizeof(float) * this->output_numel));

    // Specify the data input size used during the current inference
    auto input_dims = engine->getBindingDimensions(0);
    input_dims.d[0] = input_batch;

    this->execution_context->setBindingDimensions(0, input_dims);

    this->bindings[0] = this->input_data_device;
    this->bindings[1] = this->output_data_device;

    return true;
}


void YoloImpl::preprocess(const std::string& device, float* d2i, cv::Mat& image) {
    // 计算缩放比例scale，正变换矩阵i2d，逆变换矩阵d2i
    float scale_x = input_width / (float)image.cols;
    float scale_y = input_height / (float)image.rows;
    float scale = std::min(scale_x, scale_y);
    // 正变换矩阵，缩放到目标尺寸，且长边对齐，位置居中
    float i2d[6];
    i2d[0] = scale;  i2d[1] = 0;  i2d[2] = (-scale * (image.cols - 1) + input_width - 1) * 0.5;
    i2d[3] = 0;  i2d[4] = scale;  i2d[5] = (-scale * (image.rows - 1) + input_height - 1) * 0.5;

    // 用Opencv计算逆变换矩阵d2i
    cv::Mat m2x3_i2d(2, 3, CV_32F, i2d);  // image to dst(network), 2x3 matrix
    cv::Mat m2x3_d2i(2, 3, CV_32F, d2i);  // dst to image, 2x3 matrix
    cv::invertAffineTransform(m2x3_i2d, m2x3_d2i);
    if (device == "gpu") {
        float* d2i_device = nullptr;
        checkRuntime(hipMalloc(&d2i_device, 6 * sizeof(float)));
        checkRuntime(hipMemcpyAsync(d2i_device, d2i, 6 * sizeof(float), hipMemcpyHostToDevice, stream));

        uint8_t* src_device = nullptr;
        float* dst_device = input_data_device;
        size_t src_size = image.cols * image.rows * 3;
        checkRuntime(hipMalloc(&src_device, src_size * sizeof(uint8_t)));
        checkRuntime(hipMemcpyAsync(src_device, image.data, src_size, hipMemcpyHostToDevice, stream));

        warp_affine_bilinear(
            src_device, image.cols, image.rows,
            dst_device, input_width, input_height,
            114, d2i_device, this->stream
        );

        checkRuntime(hipFree(src_device));
        checkRuntime(hipFree(d2i_device));
    }
    else if (device == "cpu") {
        cv::Mat input_image(input_height, input_width, CV_8UC3);
        // Translate, scale
        cv::warpAffine(image, input_image, m2x3_i2d, input_image.size(), cv::INTER_LINEAR, cv::BORDER_CONSTANT, cv::Scalar::all(114));

        int image_area = input_image.cols * input_image.rows;
        unsigned char* pimage = input_image.data;
        float* phost_b = input_data_host + image_area * 0;
        float* phost_g = input_data_host + image_area * 1;
        float* phost_r = input_data_host + image_area * 2;
        for(int i = 0; i < image_area; ++i, pimage += 3){
            // bgr -> rgb
            *phost_r++ = pimage[0] / 255.0f;
            *phost_g++ = pimage[1] / 255.0f;
            *phost_b++ = pimage[2] / 255.0f;
        }
        checkRuntime(hipMemcpyAsync(this->input_data_device, this->input_data_host, this->input_numel * sizeof(float), hipMemcpyHostToDevice, stream));
    } else {
        printf("Device `%s` is not supported\n", device.c_str());
    }
}


void YoloImpl::detect(cv::Mat& image, std::vector<DetectBox>& allDetections) {
    float d2i[6];
    this->preprocess("gpu", d2i, image);

    bool success = execution_context->enqueueV2((void**)bindings, stream, nullptr);
    checkRuntime(hipStreamSynchronize(stream));

    this->postprocess("gpu",
                      output_numel,
                      output_numbox,
                      output_data_device,
                      num_classes,
                      d2i,
                      allDetections);
}


void YoloImpl::postprocess(const std::string& device,
                           const int& output_numel,
                           const int& output_numbox,
                           float* output_data_device,
                           const int& num_classes,
                           float* d2i,
                           std::vector<DetectBox>& allDetections) {
    if (device == "cpu") {
        postprocess_cpu(output_numel,
                        output_numbox,
                        output_data_device,
                        num_classes,
                        d2i,
                        allDetections);
    }
    else if (device == "gpu") {
        postprocess_gpu(output_numbox,
                        output_data_device,
                        num_classes,
                        d2i,
                        allDetections);        
    } else {
        printf("Device `%s` is not supported\n", device.c_str());
    }
}

void YoloImpl::postprocess_cpu(const int& output_numel,
                               const int& output_numbox,
                               float* output_data_device,
                               const int& num_classes,
                               float* d2i,
                               std::vector<DetectBox>& allDetections) {

    float* output_data_host = nullptr;
    checkRuntime(hipHostMalloc(&output_data_host, sizeof(float) * output_numel));
    checkRuntime(hipMemcpyAsync(output_data_host, output_data_device, sizeof(float) * output_numel, hipMemcpyDeviceToHost, stream));
    checkRuntime(hipStreamSynchronize(stream));

    // decode box：Restore predictions from different scales 
    // to the original input image(bbox, probability, confidence）
    std::vector<std::vector<float>> bboxes;
    float confidence_threshold = 0.25;
    float nms_threshold = 0.5;
    for(int i = 0; i < output_numbox; ++i){
        float* ptr = output_data_host + i * (num_classes + 5);
        float objness = ptr[4];
        if(objness < confidence_threshold)
            continue;

        float* pclass = ptr + 5;
        int label     = std::max_element(pclass, pclass + num_classes) - pclass;
        if (std::string(cocolabels[label]) != "person") {  // only detect and track person
            continue;
        }
        float prob    = pclass[label];
        float confidence = prob * objness;
        if(confidence < confidence_threshold)
            continue;

        // center(cx, cy), width, height
        float cx     = ptr[0];
        float cy     = ptr[1];
        float width  = ptr[2];
        float height = ptr[3];

        // bbox
        float left   = cx - width * 0.5;
        float top    = cy - height * 0.5;
        float right  = cx + width * 0.5;
        float bottom = cy + height * 0.5;

        // bbox position on image
        float image_base_left   = d2i[0] * left   + d2i[2];
        float image_base_right  = d2i[0] * right  + d2i[2];
        float image_base_top    = d2i[4] * top    + d2i[5];
        float image_base_bottom = d2i[4] * bottom + d2i[5];
        bboxes.push_back({image_base_left, image_base_top, image_base_right, image_base_bottom, confidence, (float)label});
    }

    // nms
    std::sort(bboxes.begin(), bboxes.end(), [](std::vector<float>& a, std::vector<float>& b){return a[4] > b[4];});
    std::vector<bool> remove_flags(bboxes.size());

    allDetections.clear();
    allDetections.reserve(bboxes.size());

    auto iou = [](const std::vector<float>& a, const std::vector<float>& b) {
        float cross_left   = std::max(a[0], b[0]);
        float cross_top    = std::max(a[1], b[1]);
        float cross_right  = std::min(a[2], b[2]);
        float cross_bottom = std::min(a[3], b[3]);

        float cross_area = std::max(0.0f, cross_right - cross_left) * std::max(0.0f, cross_bottom - cross_top);
        float union_area = std::max(0.0f, a[2] - a[0]) * std::max(0.0f, a[3] - a[1]) 
                        + std::max(0.0f, b[2] - b[0]) * std::max(0.0f, b[3] - b[1]) - cross_area;
        if(cross_area == 0 || union_area == 0) return 0.0f;
        return cross_area / union_area;
    };

    for(int i = 0; i < bboxes.size(); ++i) {
        if(remove_flags[i]) continue;

        auto& ibox = bboxes[i];

        float left = ibox[0];
        float top = ibox[1];
        float right = ibox[2];
        float bottom = ibox[3];
        float confidence = ibox[4];
        int class_label = ibox[5];
        DetectBox db(left, top, right, bottom, confidence, class_label);
        allDetections.emplace_back(db);
        for(int j = i + 1; j < bboxes.size(); ++j){
            if(remove_flags[j]) continue;

            auto& jbox = bboxes[j];
            if(ibox[5] == jbox[5]) {
                // class matched
                if(iou(ibox, jbox) >= nms_threshold)
                    remove_flags[j] = true;
            }
        }
    }

    allDetections.shrink_to_fit();
    checkRuntime(hipHostFree(output_data_host));
}


void YoloImpl::postprocess_gpu(const int& output_numbox,
                               float* predict_device,
                               const int& num_classes,
                               float* d2i,
                               std::vector<DetectBox>& allDetections) {
    // decode box：Restore predictions from different scales 
    // to the original input image(bbox, probability, confidence)
    float* output_device = nullptr;
    float* output_host = nullptr;
    const int max_objects = 1000;
    const int NUM_BOX_ELEMENT = 7;  // left, top, right, bottom, confidence, class, keepflag
    float confidence_threshold = 0.25;
    float nms_threshold = 0.5;

    float* d2i_device = nullptr;
    checkRuntime(hipMalloc(&d2i_device, 6 * sizeof(float)));
    checkRuntime(hipMemcpyAsync(d2i_device, d2i, 6 * sizeof(float), hipMemcpyHostToDevice, this->stream));
    // [count, box1, box2, ...]
    checkRuntime(hipMalloc(&output_device, sizeof(float) + max_objects * NUM_BOX_ELEMENT * sizeof(float)));
    // [count=0, box1, box2, ...]
    checkRuntime(hipMemset(output_device, 0, sizeof(float)));
    checkRuntime(hipHostMalloc(&output_host, sizeof(float) + max_objects * NUM_BOX_ELEMENT * sizeof(float)));

    decode_kernel_invoker(
        predict_device, output_numbox, num_classes, confidence_threshold, 
        nms_threshold, d2i_device, output_device, max_objects, NUM_BOX_ELEMENT, this->stream
    );
    checkRuntime(hipMemcpyAsync(output_host, output_device, 
        sizeof(float) + max_objects * NUM_BOX_ELEMENT * sizeof(float), 
        hipMemcpyDeviceToHost, this->stream
    ));
    checkRuntime(hipStreamSynchronize(this->stream));

    int num_boxes = min((int)output_host[0], max_objects);

    allDetections.clear();
    allDetections.reserve(num_boxes);

    for(int i = 0; i < num_boxes; ++i){
        float* ptr = output_host + 1 + NUM_BOX_ELEMENT * i;
        int keep_flag = ptr[6];
        if(keep_flag){
            // left, top, right, bottom, confidence, class
            DetectBox db(ptr[0], ptr[1], ptr[2], ptr[3], ptr[4], (int)ptr[5]);
            allDetections.emplace_back(db);
        }
    }
    allDetections.shrink_to_fit();
    checkRuntime(hipFree(d2i_device));
    checkRuntime(hipFree(output_device));
    checkRuntime(hipHostFree(output_host));
    
}

std::shared_ptr<Yolo> create_yolo(char* onnxFile,
                                  char* engineFile,
                                  bool useInt8,
                                  int input_batch,
                                  int input_channel,
                                  int input_height,
                                  int input_width,
                                  TRTLogger* logger) {
    std::shared_ptr<YoloImpl> instance(new YoloImpl(onnxFile, 
                                                    engineFile,
                                                    useInt8,
                                                    input_batch,
                                                    input_channel,
                                                    input_height,
                                                    input_width,
                                                    logger));
    if (!instance->build_model()) {
        instance.reset();
        return instance;
    }
    if (!instance->load_model()) {
        instance.reset();
        return instance;
    }
    return instance;
}